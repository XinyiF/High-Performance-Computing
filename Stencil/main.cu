#include "stencil.cuh"
#include <iostream>
using namespace std;

int main(int argc, const char *argv[]) {
  string N, r, Threads_per_block;
  if (argc > 1) {
    N = string(argv[1]);
    r = string(argv[2]);
    Threads_per_block = string(argv[3]);
  }
  unsigned int n = atoi(N.c_str());
  unsigned int R = atoi(r.c_str());
  unsigned int threads_per_block = atoi(Threads_per_block.c_str());
  float *image, *mask, *output;
  hipMallocManaged((float **)&image, n * sizeof(float));
  hipMallocManaged((float **)&output, n * sizeof(float));
  hipMallocManaged((float **)&mask, (2 * R + 1) * sizeof(float));
  for (unsigned int i = 0; i < n; i++) {
    image[i] = i;
  }
  for (unsigned int i = 0; i < 2 * R + 1; i++) {
    mask[i] = 1;
  }
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  stencil(image, mask, output, n, R, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  for (unsigned int i = 0; i < n; i++) {
    cout << output[i] << endl;
  }
  cout << "runtime is " << ms << endl;
  hipFree(image);
  hipFree(mask);
  hipFree(output);

  return 0;
}
