#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include <iostream>
using namespace std;

__global__ void stencil_kernel(const float *image, const float *mask,
                               float *output, unsigned int n, unsigned int R) {
  extern __shared__ float s[];
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int len = blockDim.x;
  float *s_image = &s[0];
  float *s_output = &s[len + 2 * R];
  float *s_mask = &s[2 * len + 2 * R];

  if (threadIdx.x < len / 2 + R) {
    if (threadIdx.x >= R)
      s_image[threadIdx.x] = image[blockIdx.x * blockDim.x + threadIdx.x - R];
    else
      s_image[threadIdx.x] = 0;
    if (len / 2 + R + threadIdx.x < R + blockDim.x)
      s_image[len / 2 + R + threadIdx.x] =
          image[blockIdx.x * blockDim.x + len / 2 + R + threadIdx.x - R];
    else
      s_image[len / 2 + R + threadIdx.x] = 0;
  }

  if (threadIdx.x < (2 * R + 1))
    s_mask[threadIdx.x] = mask[threadIdx.x];
  s_output[threadIdx.x] = 0;
  __syncthreads();
  if (i < n)
    for (int j = (int)(-R); j <= int(R); j++) {
      s_output[threadIdx.x] += s_image[threadIdx.x + j + R] * s_mask[R + j];
    }
  output[i] = s_output[threadIdx.x];
}

__host__ void stencil(const float *image, const float *mask, float *output,
                      unsigned int n, unsigned int R,
                      unsigned int threads_per_block) {

  stencil_kernel<<<(n + threads_per_block - 1) / threads_per_block,
                   threads_per_block,
                   (threads_per_block * 2 + 4 * R + 1) * sizeof(float)>>>(
      image, mask, output, n, R);
  hipDeviceSynchronize();
}
