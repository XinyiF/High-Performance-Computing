#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "KF.cuh"
#include "matrixOP.cuh"
#include "hipblas.h"
#include <math.h>
#include <iostream>


using std::cout;
int read_all_data(float* acc, float *gyro, float *gps, float *altitude,
int* applyKF, int* applyZUPT, int* applyZARU, int* applyALTITUDE);
int main(int argc, char** argv)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int n = 1;
    Filter* h_filter;
    hipMallocManaged(&h_filter, sizeof(Filter));
    Hx* h_H;
    hipMallocManaged(&h_H, sizeof(Hx));
    float* acc, * gyro, * gps, * altitude;
    int* applyKF, * applyZUPT, * applyZARU,*applyGPS, * applyALTITUDE;

////--------------------------reading data---------------------
    int data_points=read_all_data(acc, gyro, gps, altitude, applyKF, applyZUPT, applyZARU, applyALTITUDE);

    float* pos_fd, * var_fd, * pos_bk, * var_bk, * pos;
    hipMallocManaged(&pos_fd, 3 * data_points * sizeof(float));
    hipMallocManaged(&pos_bk, 3 * data_points * sizeof(float));
    hipMallocManaged(&pos, 3 * data_points * sizeof(float));
    hipMallocManaged(&var_fd, 3 * data_points * sizeof(float));
    hipMallocManaged(&var_bk, 3 * data_points * sizeof(float));
///------------------------forward integral------------------------
    hipEventRecord(start);

    init_KF(h_filter, h_H, handle);
    // print_KF(h_filter, h_H);

    hipMallocManaged(&h_filter, sizeof(Filter));
    forward_integrate<<<1,1>>>(handle, h_filter, h_H, data_points, acc, gyro, gps, altitude, applyKF, applyZUPT, applyZARU, applyGPS, applyALTITUDE, pos_fd, var_fd);
    hipDeviceSynchronize();
///------------------------backward integral------------------------
    init_KF(h_filter, h_H, handle);
    backward_integrate << <1, 1 >> > (handle, h_filter, h_H, data_points, acc, gyro, gps, altitude, applyKF, applyZUPT, applyZARU, applyGPS, applyALTITUDE, pos_bk, var_bk);
    hipDeviceSynchronize();
///------------------------smoother------------------------
    pos_kernel <<<(n + 1023) / 1024, 1024 >> > (pos_fd, pos_bk, var_fd, var_bk, pos, data_points);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << time << "\n";

    hipblasDestroy(handle);
    hipFree(acc);
    hipFree(gyro);
    hipFree(gps);
    hipFree(altitude);
    hipFree(applyKF);
    hipFree(applyZUPT);
    hipFree(applyZARU);
    hipFree(applyGPS);
    hipFree(applyALTITUDE);
    hipFree(pos_fd);
    hipFree(pos_bk);
    hipFree(pos);
    hipFree(var_fd);
    hipFree(var_bk);
    return 0;
}

int read_all_data(float* acc, float* gyro, float* gps, float* altitude,
    int* applyKF, int* applyZUPT, int* applyZARU, int* applyALTITUDE)
{
    string filename;
    int col,lines,res;
    int number;
    filename = "applyKF.txt"; col = 1;
    lines = read_data<int>(filename, NULL, 0, col);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, col);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";
    number = lines;

    filename = "applyZUPT.txt"; col = 1;
    lines = read_data<int>(filename, NULL, 0, 3);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, 3);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";

    filename = "applyZARU.txt"; col = 1;
    lines = read_data<int>(filename, NULL, 0, 3);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, 3);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";

    filename = "applyALTITUDE.txt"; col = 1;
    lines = read_data<int>(filename, NULL, 0, 3);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, 3);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";

    filename = "acce.txt"; col = 3;
    lines = read_data<int>(filename, NULL, 0, 3);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, 3);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";

    filename = "gyro.txt"; col = 3;
    lines = read_data<int>(filename, NULL, 0, 3);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, 3);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";

    filename = "gps.txt"; col = 3;
    lines = read_data<int>(filename, NULL, 0, 3);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, 3);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";

    filename = "altitude.txt"; col = 1;
    lines = read_data<int>(filename, NULL, 0, 3);
    hipMallocManaged(&applyKF, lines * col * sizeof(int));
    res = read_data<int>(filename, applyKF, lines, 3);
    if (res != 1)
        cout << "read " << filename << " failed\n";
    else
        cout << "File " << filename << " loaded \n";

    return number;
}