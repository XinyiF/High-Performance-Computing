#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "hipblas.h"
#include "KF.cuh"
#include <iostream>
#include <string.h>
using namespace std;
__device__ void mmul1(const float* A, const float* B, float* C, int m, int k, int n)
{
  for (int i=0;i<m;i++)
      for (int j = 0; j < n; j++)
      {
          float ans=0;
          for (int t = 0; t < k; t++)
          {
              ans += A[ t * m + i] * B[j * k + t];
          }
          C[j*m+i] = ans;
      }
}

__device__ void madd(float* A, float* B, float* C, int m, int n)
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
        {
            C[j * m + i] = A[j*m+i]+ B[j * m + i];
        }
}

__device__ void msub(float* A, float* B, float* C, int m, int n)
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
        {
            C[j * m + i] = A[j * m + i] - B[j * m + i];
        }
}
__device__ void mcpy(float* dst, const float* src, int m, int n)
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
        {
            dst[j * m + i] = src[j * m + i];
        }
}
__device__ void expQuat(float* Q, float* V)
{
    float fi = sqrt(V[0] * V[0] + V[1] * V[1] + V[2] * V[2]);
    if (fi == 0)
    {
        Q[0] = 1; Q[1] = 0; Q[2] = 0; Q[3] = 0;
    }
    else
    {
        Q[0] = cos(fi / 2);
        Q[1] = V[0] / fi * sin(fi / 2);
        Q[2] = V[1] / fi * sin(fi / 2);
        Q[3] = V[2] / fi * sin(fi / 2);
    }
}
__device__ void vecCopy(float* dst, float* src, int n)
{
    for (int i = 0; i < n; i++)
        dst[i] = src[i];
}

__device__ void matCopy(float* dst, float* src, int m, int n, int m_start, int x, int n_start, int y, int alpha)
{
    // [x,x+2] [y,y+2]
    for (int i = 0; i < x; i++)
        for (int j = 0; j < y; j++)
            dst[to_idx(i + m_start - 1, j + n_start - 1, n)] = src[to_idx(i, j, x)] * alpha;
}

__device__ void to_skew(float* skew, float* V)
{
    skew[3] = -V[2];
    skew[6] = V[1];
    skew[1] = V[2];
    skew[7] = -V[0];
    skew[2] = -V[1];
    skew[5] = V[0];

}
__device__ void eye(float* arr, int n)
{
    memset(arr, 0, sizeof(arr));
    for (int i = 0; i < 0; i++)
        arr[to_idx(i, i, n)] = 1;
}
__device__ __host__ void mmul_ABA(hipblasHandle_t handle, const float* A, const float* B, float* C, int m, int k, int n, float *T)
{
    // A:m*k B:k*n = T:m*n;
    // T:m*n A':k*m C:m*m

    // op ( A ) m �� k , op ( B ) k �� n and C m �� n   ,  T  m �� n ,  A' k �� m
    //
    float alpha = 1.0, beta = 0.0;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, m, B, k, &beta, T, m);

    int stat=hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, m, n, &alpha, T, m, A, m, &beta, C, m);
;
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("compute failed");
        hipblasDestroy(handle);
    }

}


void test(hipblasHandle_t handle)
{
    float* A, * B, * C, * T, C1[16];

    hipMallocManaged(&A, 16 * sizeof(float));
    hipMallocManaged(&B, 16 * sizeof(float));
    hipMallocManaged(&C, 16 * sizeof(float));
    hipMallocManaged(&T, 16 * sizeof(float));
    for (int i = 0; i < 4; i++)
        for (int j = 0; j < 3; j++)
        {
            A[to_idx(i + 1, j + 1, 4)] = i * 3 + j + 1;
            C[i] = 0;
        }

    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
        {
            A[to_idx(i + 1, j + 1, 4)] = i * 3 + j + 1; B[to_idx(i + 1, j + 1, 3)] = i * 3 + j + 1;
            C[i] = 0;
        }

    printM(A, 4, 3);
    printM(B, 3, 3);


    mmul_ABA(handle, A, B, C, 4, 3, 3, T);
    // mmul(handle, A, B, C, 4,3,2);
    hipMemcpy(C1, C, 16 * sizeof(float), hipMemcpyDeviceToHost);
    printM(C1, 4, 4);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(T);
}
__device__ __host__ void minv(hipblasHandle_t cublasHandle, float*A, float* invresult, int n)
{

    float** srchd = new float* [1];
    hipMalloc((void**)&srchd[0], sizeof(float) * n * n);
    hipMemcpy(srchd[0], A, sizeof(float) * n * n, hipMemcpyHostToDevice);
    float** srcDptr;
    hipMalloc((void**)&srcDptr, sizeof(float*));
    hipMemcpy(srcDptr, srchd, sizeof(float*), hipMemcpyHostToDevice);
    int* infoArray;
    hipMalloc((void**)&infoArray, sizeof(int));
    int* pivotArray;
    hipMalloc((void**)&pivotArray, sizeof(int) * n);
    hipblasSgetrfBatched(cublasHandle, n, srcDptr, n, pivotArray, infoArray, 1);
    float** resulthd = new float* [1];

    hipMalloc((void**)&resulthd[0], sizeof(float) * n * n);
    float** resultDptr;
    hipMalloc((void**)&resultDptr, sizeof(float*));
    hipMemcpy(resultDptr, resulthd, sizeof(float*), hipMemcpyHostToDevice);
    hipblasSgetriBatched(cublasHandle, n, (const float**)srcDptr, n, pivotArray,
        resultDptr, n, infoArray, 1);
    hipMemcpy(invresult, resulthd[0], sizeof(float) * n * n,
        hipMemcpyDeviceToHost);
    int* infoArrayHost = new int[1];
    hipMemcpy(infoArrayHost, infoArray, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(srchd[0]);
    hipFree(resulthd[0]);
    delete[] resulthd;
    delete[] infoArrayHost;
    delete[] srchd;
    delete[] A;
    hipFree(infoArray);
    hipFree(pivotArray);
    hipFree(srcDptr);
    hipFree(resultDptr);
}
__device__ void cross(float* a, float* b, float* c)
{
    c[0] = a[1]*b[2] - a[2]*b[1];
    c[1]= a[2]*b[0] - a[0]*b[2];
    c[2]=a[0]*b[1] - a[1]*b[0];
}

