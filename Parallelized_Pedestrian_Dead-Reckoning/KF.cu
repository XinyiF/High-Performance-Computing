#include "hip/hip_runtime.h"
//Yisen 5/2020
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "hipblas.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <stdio.h>
#include <cmath>
#include "KF.cuh"
#include "matrixOP.cuh"

using namespace std;
#define dt 0.005

//initialize matrix in Kalman filter
__host__ void init_KF(Filter* F, Hx* H, hipblasHandle_t handle)
{
	F->PKF[to_idx(1, 1, 15)] = 0.1;
	F->PKF[to_idx(2, 2, 15)] = 0.1;
	F->PKF[to_idx(3, 3, 15)] = 0.1;
	F->PKF[to_idx(9, 9, 15)] = 0.5;

	F->QKF[to_idx(1, 1, 12)] = ((1e-3 * dt) * (1e-3 * dt) + (4.5e-3) * (4.5e-3) * dt);
	F->QKF[to_idx(2, 2, 12)] = ((1e-3 * dt) * (1e-3 * dt) + (4.5e-3) * (4.5e-3) * dt);
	F->QKF[to_idx(3, 3, 12)] = ((1e-3 * dt) * (1e-3 * dt) + (4.5e-3) * (4.5e-3) * dt);
	F->QKF[to_idx(4, 4, 12)] = (pow( (2.5*1e-3 * pi / 180 * dt), 2) + pow((7.4e-3 * pi / 180), 2) * dt) * 20;
	F->QKF[to_idx(5, 5, 12)] = (pow( (2.5*1e-3 * pi / 180 * dt), 2) + pow((7.4e-3 * pi / 180), 2) * dt) * 20;
	F->QKF[to_idx(6, 6, 12)] = (pow( (2.5*1e-3 * pi / 180 * dt), 2) + pow((7.4e-3 * pi / 180), 2) * dt) * 20;
	F->QKF[to_idx(7, 7, 12)] = pow((6e-5), 2) * dt;
	F->QKF[to_idx(8, 8, 12)] = pow((6e-5), 2) * dt;
	F->QKF[to_idx(9, 9, 12)] = pow((6e-5), 2) * dt;
	F->QKF[to_idx(10, 10, 12)] = pow((3e-4 * pi / 180), 2) * dt;
	F->QKF[to_idx(11, 11, 12)] = pow((3e-4 * pi / 180), 2) * dt;
	F->QKF[to_idx(12, 12, 12)] = pow((3e-4 * pi / 180), 2) * dt;

	F->RKF[ZUPT] = 1e-3;
	F->RKF[ZARU] = 5e-5;
	F->RKF[GPS] = 1e-1;
	F->RKF[ALTITUDE] = 1e-1;
	F->RKF[ELE] = 1e-1;

	F->Fi[to_idx(4, 1, 15)] = 1;
	F->Fi[to_idx(5, 2, 15)] = 1;
	F->Fi[to_idx(6, 3, 15)] = 1;
	F->Fi[to_idx(7, 4, 15)] = 1;
	F->Fi[to_idx(8, 5, 15)] = 1;
	F->Fi[to_idx(9, 6, 15)] = 1;
	F->Fi[to_idx(10, 7, 15)] = 1;
	F->Fi[to_idx(11, 8, 15)] = 1;
	F->Fi[to_idx(12, 9, 15)] = 1;
	F->Fi[to_idx(13, 10, 15)] = 1;
	F->Fi[to_idx(14, 11, 15)] = 1;
	F->Fi[to_idx(15, 12, 15)] = 1;

	float* T,*T1;
	hipMallocManaged(&T, 15 * 15 * sizeof(float));
	hipMallocManaged(&T1, 15 * 15 * sizeof(float));
	mmul_ABA(handle, F->Fi, F->QKF,T1 , 15, 12, 12, T);	//// F->QKFT= Fi*QKFT*Fi';
	hipFree(T);
	hipMemcpy(F->QKFT, T1, 15 * 15 * sizeof(float), hipMemcpyDefault);
	
	int H_idx[16] = {0, 1 , 3 , 3 , 3 , 4 , 6 , 6 , 3 , 3 , 6 , 5 , 6 , 6 , 9 , 8};
	for (int i = 1; i < 16; i++)
	{
		H->H_size[i * 2] = H_idx[i];
		H->H_size[i * 2+1] = 16;
	}
	H->H_s[1] = 0;
	for (int i = 2; i < 16; i++)
	{
		H->H_s[i] = H->H_s[i-1] +H_idx[i-1]*16;
	}
	///////////////---------------------------------/////////////////////////////////////
	if (1 == 1)
	{
		float* Hx;
		Hx = &(H->Hx[(int)H->H_s[1]]);
		Hx[to_idx(1, 3, 1)] = 1;

		Hx = &(H->Hx[(int)H->H_s[2]]);
		Hx[to_idx(1, 1, 3)] = 1; Hx[to_idx(2, 2, 3)] = 1; Hx[to_idx(3, 3, 3)] = 1;

		Hx = &(H->Hx[(int)H->H_s[3]]);
		Hx[to_idx(1, 1, 3)] = 1; Hx[to_idx(2, 2, 3)] = 1; Hx[to_idx(3, 3, 3)] = 1;

		Hx = &(H->Hx[(int)H->H_s[4]]);
		Hx[to_idx(1, 14, 3)] = 1; Hx[to_idx(2, 15, 3)] = 1; Hx[to_idx(3, 16, 3)] = 1;

		Hx = &(H->Hx[(int)H->H_s[5]]);
		Hx[to_idx(1, 3, 4)] = 1; Hx[to_idx(2, 14, 4)] = 1; Hx[to_idx(3, 15, 4)] = 1; Hx[to_idx(4, 16, 4)] = 1;

		Hx = &(H->Hx[(int)H->H_s[6]]);
		Hx[to_idx(1, 1, 6)] = 1; Hx[to_idx(2, 2, 6)] = 1; Hx[to_idx(3, 3, 6)] = 1; 
		Hx[to_idx(4, 14, 6)] = 1; Hx[to_idx(5, 15, 6)] = 1; Hx[to_idx(6, 16, 6)] = 1;

		Hx = &(H->Hx[(int)H->H_s[7]]);
		Hx[to_idx(1, 1, 6)] = 1; Hx[to_idx(2, 2, 6)] = 1; Hx[to_idx(3, 3, 6)] = 1;
		Hx[to_idx(4, 14, 6)] = 1; Hx[to_idx(5, 15, 6)] = 1; Hx[to_idx(6, 16, 6)] = 1;

		Hx = &(H->Hx[(int)H->H_s[8]]);
		Hx[to_idx(1, 4, 3)] = 1; Hx[to_idx(2, 5, 3)] = 1; Hx[to_idx(3, 6, 3)] = 1;

		Hx = &(H->Hx[(int)H->H_s[9]]);
		Hx[to_idx(1, 3, 3)] = 1; Hx[to_idx(2, 4, 3)] = 1; Hx[to_idx(3, 5, 3)] = 1;

		Hx = &(H->Hx[(int)H->H_s[10]]);
		Hx[to_idx(1, 1, 6)] = 1; Hx[to_idx(2, 2, 6)] = 1; Hx[to_idx(3, 3, 6)] = 1; 
		Hx[to_idx(4, 4, 6)] = 1; Hx[to_idx(5, 5, 6)] = 1; Hx[to_idx(6, 6, 6)] = 1;

		Hx = &(H->Hx[(int)H->H_s[11]]);
		Hx[to_idx(1, 1, 5)] = 1; Hx[to_idx(2, 2, 5)] = 1; Hx[to_idx(3, 3, 5)] = 1; 
		Hx[to_idx(4, 4, 5)] = 1; Hx[to_idx(5, 5, 5)] = 1;

		Hx = &(H->Hx[(int)H->H_s[12]]);
		Hx[to_idx(1, 4, 6)] = 1; Hx[to_idx(2, 5, 6)] = 1; Hx[to_idx(3, 6, 6)] = 1; 
		Hx[to_idx(4, 14, 6)] = 1; Hx[to_idx(5, 15, 6)] = 1; Hx[to_idx(6, 16, 6)] = 1;

		Hx = &(H->Hx[(int)H->H_s[13]]);
		Hx[to_idx(1, 3, 6)] = 1; Hx[to_idx(2, 4, 6)] = 1; Hx[to_idx(3, 5, 6)] = 1;
		Hx[to_idx(4, 14, 6)] = 1; Hx[to_idx(5, 15, 6)] = 1; Hx[to_idx(6, 16, 6)] = 1;

		Hx = &(H->Hx[(int)H->H_s[14]]);
		Hx[to_idx(1, 1, 9)] = 1; Hx[to_idx(2, 2, 9)] = 1; Hx[to_idx(3, 3, 9)] = 1; 
		Hx[to_idx(4, 4, 9)] = 1; Hx[to_idx(5, 5, 9)] = 1; Hx[to_idx(6, 6, 9)] = 1;
		Hx[to_idx(7, 14, 9)] = 1; Hx[to_idx(8, 15, 9)] = 1; Hx[to_idx(9, 16, 9)] = 1;

		Hx = &(H->Hx[(int)H->H_s[15]]);
		Hx[to_idx(1, 1, 8)] = 1; Hx[to_idx(2, 2, 8)] = 1; Hx[to_idx(3, 3, 8)] = 1; 
		Hx[to_idx(4, 4, 8)] = 1; Hx[to_idx(5, 5, 8)] = 1;
		Hx[to_idx(6, 14, 8)] = 1; Hx[to_idx(7, 15, 8)] = 1; Hx[to_idx(8, 16, 8)] = 1;
	}

	int M_idx[16] = { 0,1 , 3 , 3 , 3 , 4 , 6 , 6 , 3 , 3 , 6 , 5 , 6 , 6 , 9 , 8 };
	for (int i = 1; i < 16; i++)
	{
		H->M_size[i * 2] = M_idx[i];
		H->M_size[i * 2 + 1] = M_idx[i];
	}
	H->M_s[1] = 0;
	for (int i = 2; i < 16; i++)
	{
		H->M_s[i] = H->M_s[i - 1] + M_idx[i - 1]* M_idx[i - 1];
	}
	if (1 == 1)
	{
		float* Mx;
		Mx = &(H->Mx[(int)H->M_s[1]]);
		Mx[to_idx(1, 1, 1)] = 0.1;

		Mx = &(H->Mx[(int)H->M_s[2]]);
		Mx[to_idx(1, 1, 3)] = 0.1; Mx[to_idx(2, 2,3)] = 0.1; Mx[to_idx(3, 3, 3)] = 0.1;

		Mx = &(H->Mx[(int)H->M_s[3]]);
		Mx[to_idx(1, 1, 3)] = 0.1; Mx[to_idx(2, 2,3)] = 0.1; Mx[to_idx(3, 3,3)] = 0.1;

		Mx = &(H->Mx[(int)H->M_s[4]]);
		Mx[to_idx(1, 1, 3)] = 5e-5; Mx[to_idx(2, 2,3)] = 5e-5; Mx[to_idx(3, 3, 3)] = 5e-5;

		Mx = &(H->Mx[(int)H->M_s[5]]);
		Mx[to_idx(1, 1, 4)] = 0.1; Mx[to_idx(2, 2, 4)] = 5e-5; Mx[to_idx(3, 3, 4)] = 5e-5; Mx[to_idx(4, 4, 4)] = 5e-5;

		Mx = &(H->Mx[(int)H->M_s[6]]);
		Mx[to_idx(1, 1, 6)] = 0.1; Mx[to_idx(2, 2,6)] = 0.1; Mx[to_idx(3, 3, 6)] = 0.1; 
		Mx[to_idx(4, 4, 6)] = 5e-5; Mx[to_idx(5, 5, 6)] = 5e-5; Mx[to_idx(6, 6, 6)] = 5e-5;
		
		Mx = &(H->Mx[(int)H->M_s[7]]);
		Mx[to_idx(1, 1, 6)] = 0.1; Mx[to_idx(2, 2, 6)] = 0.1; Mx[to_idx(3, 3, 6)] = 0.1;
		Mx[to_idx(4, 4, 6)] = 5e-5; Mx[to_idx(5, 5, 6)] = 5e-5; Mx[to_idx(6, 6, 6)] = 5e-5;
		
		Mx = &(H->Mx[(int)H->M_s[8]]);
		Mx[to_idx(1, 1, 3)] = 1e-3; Mx[to_idx(2, 2, 3)] = 1e-3; Mx[to_idx(3, 3, 3)] = 1e-3;

		Mx = &(H->Mx[(int)H->M_s[9]]);
		Mx[to_idx(1, 1, 3)] = 1e-1; Mx[to_idx(2, 2, 3)] = 1e-3; Mx[to_idx(3, 3, 3)] = 1e-3;

		Mx = &(H->Mx[(int)H->M_s[10]]);
		Mx[to_idx(1, 1, 6)] = 0.1; Mx[to_idx(2, 2, 6)] = 0.1; Mx[to_idx(3, 3, 6)] = 0.1;
		Mx[to_idx(4, 4, 6)] = 1e-3; Mx[to_idx(5, 5, 6)] = 1e-3; Mx[to_idx(6, 6, 6)] = 1e-3;

		Mx = &(H->Mx[(int)H->M_s[11]]);
		Mx[to_idx(1, 1, 5)] = 0.1; Mx[to_idx(2, 2, 5)] = 0.1; Mx[to_idx(3, 3, 5)] = 0.1;
		Mx[to_idx(4, 4, 5)] = 1e-3; Mx[to_idx(5, 5, 5)] = 1e-3; 
		
		Mx = &(H->Mx[(int)H->M_s[12]]);
		Mx[to_idx(1, 1, 6)] = 1e-3; Mx[to_idx(2, 2, 6)] = 1e-3; Mx[to_idx(3, 3, 6)] = 1e-3;
		Mx[to_idx(4, 4, 6)] = 5e-5; Mx[to_idx(5, 5, 6)] = 5e-5; Mx[to_idx(6, 6, 6)] = 5e-5;

		Mx = &(H->Mx[(int)H->M_s[13]]);
		Mx[to_idx(1, 1, 6)] = 0.1; Mx[to_idx(2, 2, 6)] = 1e-3; Mx[to_idx(3, 3, 6)] = 1e-3;
		Mx[to_idx(4, 4, 6)] = 5e-5; Mx[to_idx(5, 5, 6)] = 5e-5; Mx[to_idx(6, 6, 6)] = 5e-5;
		
		Mx = &(H->Mx[(int)H->M_s[14]]);
		Mx[to_idx(1, 1, 9)] = 0.1; Mx[to_idx(2, 2, 9)] = 0.1; Mx[to_idx(3, 3, 9)] = 0.1;
		Mx[to_idx(4, 4, 9)] = 1e-3; Mx[to_idx(5, 5, 9)] = 1e-3; Mx[to_idx(6, 6, 9)] = 1e-3;
		Mx[to_idx(7, 7, 9)] = 5e-5; Mx[to_idx(8, 8, 9)] = 5e-5; Mx[to_idx(9, 9, 9)] = 5e-5;

		Mx = &(H->Mx[(int)H->M_s[15]]);
		Mx[to_idx(1, 1, 8)] = 0.1; Mx[to_idx(2, 2, 8)] = 0.1; Mx[to_idx(3, 3, 8)] = 0.1;
		Mx[to_idx(4, 4, 8)] = 1e-3; Mx[to_idx(5, 5, 8)] = 1e-3;
		Mx[to_idx(6, 6, 8)] = 5e-5; Mx[to_idx(7, 7, 8)] = 5e-5; Mx[to_idx(8, 8, 8)] = 5e-5;
	}
}

__device__ void integrate(float* acc, float* gyro, float* P,
	float* V, float* Q, float* ABiasold, float* GBiasold)
{
	float RM[9];
	float a = Q[0]; float b = Q[1]; float c = Q[2]; float d = Q[3];
// Quaternion to Rotation matrix
	RM[0] = a * a + b * b - c * c - d * d;
	RM[3] = 2 * b * c - 2 * a * d;
	RM[6] = 2 * b * d + 2 * a * c;
	RM[1] = 2 * b * c + 2 * a * d;
	RM[4] = a * a - b * b + c * c - d * d;
	RM[7] = 2 * c * d - 2 * a * b;
	RM[2] = 2 * b * d - 2 * a * c;
	RM[5] = 2 * c * d + 2 * a * b;
	RM[8] = a * a - b * b - c * c + d * d;
	float vec1[3], vec2[3];
	vec1[0] = acc[0] - ABiasold[0];
	vec1[1] = acc[1] - ABiasold[1];
	vec1[2] = acc[2] - ABiasold[2];
	mmul1(RM, vec1, vec2, 3, 3, 1);
	vec1[0] = P[0] + V[0] * dt + 0.5 * (vec2[0] + 0) * dt * dt;
	vec1[1] = P[1] + V[1] * dt + 0.5 * (vec2[1] + 0) * dt * dt;
	vec1[2] = P[2] + V[2] * dt + 0.5 * (vec2[2] + g) * dt * dt;
	P[0] = vec1[0]; P[1] = vec1[1]; P[2] = vec1[2];

	vec1[0] = V[0] + (vec2[0] + 0) * dt;
	vec1[1] = V[1] + (vec2[1] + 0) * dt;
	vec1[2] = V[2] + (vec2[2] + g) * dt;
	V[0] = vec1[0]; V[1] = vec1[1]; V[2] = vec1[2];
	float Qe[4];
	vec1[0] = (gyro[0] - GBiasold[0]) * dt;
	vec1[1] = (gyro[1] - GBiasold[1]) * dt;
	vec1[2] = (gyro[2] - GBiasold[2]) * dt;
	expQuat(Qe, vec1);
	// Q*Qe
	a = Q[0] * Qe[0] + Q[1] * Qe[1] - Q[2] * Qe[2] - Q[3] * Qe[3];
	b = Q[0] * Qe[1] + Q[1] * Qe[0] + Q[2] * Qe[3] - Q[3] * Qe[2];
	c = Q[0] * Qe[2] - Q[1] * Qe[3] + Q[2] * Qe[0] + Q[3] * Qe[1];
	d = Q[0] * Qe[3] + Q[1] * Qe[2] - Q[2] * Qe[1] + Q[3] * Qe[0];

	Q[0] = a; Q[1] = b; Q[2] = c; Q[3] = d;


}

//get observation and measurement
void getHR(float* est, float* obs, int applyKFZUPT, int applyKFZARU, int applyKFGPS, int elevator, float* P, float* V, float* GBias, float* y)
{
	//y = [3]+[3]+[3]
	int n = 0;
	if (elevator == 1)
	{
		n = 3;
		vecCopy(est, P, 3);
		vecCopy(obs, y, 3);
	}
	if (applyKFGPS == 1)
	{
		n = 3;
		vecCopy(est, P, 3);
		vecCopy(obs, &y[1], 3);
	}
	if (applyKFZUPT == 1)
	{
		if (elevator == 1)
		{
			est[n] = V[0]; est[n + 1] = V[1];
			obs[n] = 0; obs[n + 1] = 0;
			n = 5;
		}
		else
		{
			est[n] = V[0]; est[n + 1] = V[n + 2]; est[n + 3] = V[2];
			obs[n] = 0; obs[n + 1] = 0; obs[n + 2] = 0;
			n = 6;
		}
	}
	if (applyKFZARU == 1)
	{
		est[n] = GBias[0]; est[n + 1] = GBias[1]; est[n + 2] = GBias[2];
		obs[n] = y[4]; obs[n + 1] = y[5]; obs[n + 2] = y[6];
	}
}

//quaternion to 3D rotation matrix 
__device__ void QtoRM(float* Q,float *RM)
{
	float a = Q[0]; float b = Q[1]; float c = Q[2]; float d = Q[3];
	RM[0] = a * a + b * b - c * c - d * d;
	RM[3] = 2 * b * c - 2 * a * d;
	RM[6] = 2 * b * d + 2 * a * c;
	RM[1] = 2 * b * c + 2 * a * d;
	RM[4] = a * a - b * b + c * c - d * d;
	RM[7] = 2 * c * d - 2 * a * b;
	RM[2] = 2 * b * d - 2 * a * c;
	RM[5] = 2 * c * d + 2 * a * b;
	RM[8] = a * a - b * b - c * c + d * d;
}

//core of Kalman filter
__device__ void ESKF(Filter* KF, Hx* H, float *y,float *acce, float *P, float *V, float* Q, 
	       float* ABias, float* GBias, int ZUPT1, int ZARU1, int GPS1, int elevator1,float *Fx,float *Xx, hipblasHandle_t handle)
{
	float est[9];
	float obs[9];
	getHR(est,obs,ZUPT1,ZARU1,GPS1,elevator1,P,V,GBias,y);

	int n= elevator1 + GPS1 * 2 + ZARU1 * 4 + ZUPT1 * 8;
	float RM[9];
	QtoRM(Q, RM);

	float A[3];
	A[0] = acce[0] - ABias[0];
	A[1] = acce[1] - ABias[1];
	A[2] = acce[2] - ABias[2];
	
	float vec[3];
	float skew[9]; //col major

	mmul1(RM,A,vec,3,1,1);
	to_skew(skew, vec);

	matCopy(Fx, skew, 15, 15, 4,3, 7,3, -dt);
	matCopy(Fx, RM, 15, 15, 4, 3,10,3, -dt);
	matCopy(Fx, RM, 15, 15, 10,3, 10,3, -dt);

	float T[15 * 15],T1[15*15]; 
	mmul_ABA(handle, Fx, KF->PKF, T, 15, 15, 15, T1);
	madd(T, KF->QKFT, KF->PKF,15, 15);

	float R[4 * 3];
	R[0] = -Q[1]; R[4] = -Q[2]; R[8] = -Q[3];
	R[1] = Q[0]; R[5] = -Q[3]; R[9] = -Q[2];
	R[2] = -Q[3]; R[6] = -Q[0]; R[10] = -Q[1];
	R[3] = Q[2]; R[7] = -Q[1]; R[11] = -Q[0];

	int sizeH[2];
	sizeH[0] = H->H_size[2 * n]; sizeH[1] = H->H_size[2 * n+1];
	float HH[9*15];
	int indexH = H->H_s[n];
	mmul1(&(H->Hx[indexH]), Xx, HH, sizeH[0], sizeH[1],15);
	int size = sizeH[0];
	float K[9 * 9], ansK[9*9],tK[9*9],invK[9*9];
	// HH size*15   ,   PKF: 15*15
	mmul_ABA(handle, HH, KF->PKF, ansK, size, 15, 15, tK);
	int indexM = H->M_s[2 * n];
	madd(ansK, &(H->Mx[indexM]), tK, size, size);
	minv(tK, invK, size, size);
	float alpha = 1.0, beta = 0.0;
	//PKF 15*15 H': 15*size
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 15, size, 15, &alpha, KF->PKF, 15, HH, 15, &beta, tK, 15);
	// tK 15*size inv:size*size
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, size, size, &alpha, tK, 15, invK, size, &beta, K, size);
	float deltax[15];
	float dif[9];
	for (int i = 0; i < size; i++)
		dif[i] = obs[i] - est[i];
	//K:15*size
	mmul1(K, dif, deltax, 15, 1, size);
	matCopy(KF->dP, deltax, 3, 1, 1, 3, 1, 3, 1);
	matCopy(KF->dV, &deltax[3], 3, 1, 1, 3, 1, 3, 1);
	matCopy(KF->dtheta, &deltax[6], 3, 1, 1, 3, 1, 3, 1);
	matCopy(KF->dABias, &deltax[9], 3, 1, 1, 3, 1, 3, 1);
	matCopy(KF->dGBias, &deltax[12], 3, 1, 1, 3, 1, 3, 1);

	eye(T, 15);
	// K: 15*size HH:size*15
	mmul1(K, HH, T1,15,size,15);
	msub(T, T1, T, 15, 15);
	mmul1(T1, KF->PKF, T, 15, 15, 15);
	mcpy(KF->PKF, T, 15, 15);
	

}

//initialize quaternion
__device__ void init_Q(float *acce,float* Q)
{
	float grav = sqrt(acce[0] * acce[0] + acce[1] * acce[1] + acce[2] * acce[2]);
	float zx = acce[0] / grav;
	float zy = acce[1] / grav;
	float zz = acce[2] / grav;
	float xx, xy, xz;
	if (zx != 0)
	{
		xx = 0;
		xy = sqrt(1 / (1 + (zy * zy) / (zz * zz)));
		xz = -xy * zy / zz;
	}
	else
	{
		if (zy != 0)
		{
			xy = 0;
			xx = sqrt(1 / (1 + (zx * zx) / (zz * zz)));
			xz = -xx * zx / zz;
		}
		else
		{
			xz = 0;
			xx = sqrt(1 / (1 + (zx * zx) / (zy * zy)));
			xy = -xx * zx / zy;
		}
	}
	float v1[3] = { zx, zy, zz };
	float v2[3] = { zx, zy, zz };
	float y[3];
	cross(v1, v2,y);
	float R[9];

	R[0] = xx; R[3] = xy;  R[6] = xz;
	R[1] = y[0]; R[4] = y[1];  R[7] = y[2];
	R[2] = zx; R[5] = zy; R[8] = zz;
}
void printM(const float* a, int x, int y)
{
	float* arr;
	arr = (float*)malloc(x*y*sizeof(float));
	hipMemcpy(arr, a, x*y * sizeof(float), hipMemcpyDefault);
	for (int i = 0; i < x; i++)
	{
		for (int j = 0; j < y; j++)
			cout<<setw(7)<<arr[j * x + i]<<" ";
		printf("\n");
	}
	printf("\n");
	free(arr);
}

//used for check result
void print_KF(Filter* F, Hx* H)
{
	//cout << "PKF:\n";	printM(F->PKF, 15, 15);
	//cout << "QKF:\n";	printM(F->QKF, 12, 12);
	//cout << "Fi:\n";	printM(F->Fi, 15, 12);
	printf("QKFT:\n");	printM(F->QKFT, 15, 15);
	for (int i = 1; i < 16; i++)
	{
	//	cout << "Hx" << i << ":\n";
	//	cout << H->H_size[i * 2] << " " << H->H_size[i * 2 + 1] << "\n";
	//	printM(&(H->Hx[(H->H_s[i])]), H->H_size[i * 2], H->H_size[i * 2 + 1]);
		cout << "Mx" << i << ":\n";
		cout << H->M_size[i * 2] << " " << H->M_size[i * 2 + 1] << "\n";
		printM(&(H->Mx[(H->M_s[i])]), H->M_size[i * 2], H->M_size[i * 2 + 1]);
	}
}

//correct results based on output of Kalman filter
__device__ void correct(float *P, float *V, float *Q,float* ABias,float* GBias, Filter *KF)
{
	P[0] += KF->dP[0];
	P[1] += KF->dP[1];
	P[2] += KF->dP[2];

	V[0] += KF->dV[0];
	V[1] += KF->dV[1];
	V[2] += KF->dV[2];

	float tQ[4];
	expQuat(tQ, KF->dtheta);

	float a = Q[0] * tQ[0] + Q[1] * tQ[1] - Q[2] * tQ[2] - Q[3] * tQ[3];
	float b = Q[0] * tQ[1] + Q[1] * tQ[0] + Q[2] * tQ[3] - Q[3] * tQ[2];
	float c = Q[0] * tQ[2] - Q[1] * tQ[3] + Q[2] * tQ[0] + Q[3] * tQ[1];
	float d = Q[0] * tQ[3] + Q[1] * tQ[2] - Q[2] * tQ[1] + Q[3] * tQ[0];
	Q[0] = a; Q[1] = b; Q[2] = c; Q[3] = d;

	ABias[0] += KF->dABias[0];
	ABias[1] += KF->dABias[1];
	ABias[2] += KF->dABias[2];

	GBias[0] += KF->dGBias[0];
	GBias[1] += KF->dGBias[1];
	GBias[2] += KF->dGBias[2];
}
//error propagation when no update apply
__device__ void covariancePropagate(hipblasHandle_t handle, Filter* KF, Hx* H, float *acce, 
	                    float *Q,float *ABias, float* Fx)
{
	float RM[9];
	QtoRM(Q, RM);
	float A[3];
	A[0] = acce[0] - ABias[0];
	A[1] = acce[1] - ABias[1];
	A[2] = acce[2] - ABias[2];
	float vec[3];
	float skew[9]; //col major
	mmul1(RM, A, vec, 3, 1, 1);
	to_skew(skew, vec);
	matCopy(Fx, skew, 15, 15, 4, 3, 7, 3, -dt);
	matCopy(Fx, RM, 15, 15, 4, 3, 10, 3, -dt);
	matCopy(Fx, RM, 15, 15, 10, 3, 10, 3, -dt);
	float T[15 * 15], T1[15 * 15]; 
	mmul_ABA(handle, Fx, KF->PKF, T, 15, 15, 15, T1);
	madd(T, KF->QKFT, KF->PKF, 15, 15);
}
__global__ void forward_integrate(hipblasHandle_t handle, Filter* KF, Hx* H, int n, float* acc, float* gyro, float* GPSdata,
	float* baro, int* applyKF, int* applyZUPT, int* applyZARU, int* applyGPS, int* applyAltitude, float* pos, float* var)
{
	float P[3];
	float V[4];
	float Q[4];
	float ABias[3];
	float GBias[3];
	float Fx[15 * 15];
	float Xx[15 * 15];
	float y[7];
	init_Q(gyro, Q);
	for (int i = n; i >= 0; i--)
	{
		integrate(&acc[i * 3 * n], &gyro[i * 3 * n], P, V, Q, ABias, GBias);
		y[0] = baro[i]; y[1] = acc[3 * i]; y[2] = acc[3 * i + 1]; y[3] = acc[3 * i + 2];
		y[4] = gyro[3 * i]; y[5] = gyro[3 * i + 1]; y[6] = gyro[3 * i] + 2;
		if (applyKF[i] == 1)
		{
			ESKF(KF, H, y, &acc[i * 3 * n], &P[i * 3 * n], V, Q, ABias, GBias, applyZUPT[i * n], applyZARU[i * n], applyGPS[i * n], applyAltitude[i * n], Fx, Xx, handle);
			correct(P, V, Q, ABias, GBias, KF);
		}
		else
		{
			covariancePropagate(handle, KF, H, &acc[i * 3 * n], &Q[i * 3 * n], &ABias[i * 3 * n], Fx);
		}
		pos[i * 3] = P[0];
		pos[i * 3 + 1] = P[1];
		pos[i * 3 + 2] = P[2];
		var[i * 3] = KF->PKF[to_idx(1, 1, 15)];
		var[i * 3 + 1] = KF->PKF[to_idx(2, 2, 15)];
		var[i * 3 + 2] = KF->PKF[to_idx(3, 3, 15)];
	}
}

__global__ void backward_integrate(hipblasHandle_t handle, Filter* KF, Hx* H, int n, float* acc, float* gyro, float* GPSdata,
	float* baro, int* applyKF, int* applyZUPT, int* applyZARU, int* applyGPS, int* applyAltitude, float* pos, float* var)
{
//	dt = -dt;
	float P[3];
	float V[4];
	float Q[4];
	float ABias[3];
	float GBias[3];
	float Fx[15 * 15];
	float Xx[15 * 15];
	float y[7];
	init_Q(gyro, Q);
	for (int i = n; i>=0; i--)
	{
		integrate(&acc[i * 3 * n], &gyro[i * 3 * n], P, V, Q, ABias, GBias);
		y[0] = baro[i]; y[1] = acc[3*i]; y[2] = acc[3 * i+1]; y[3] = acc[3 * i+2];
		y[4] = gyro[3*i]; y[5] = gyro[3 * i+1]; y[6] = gyro[3 * i]+2;
		if (applyKF[i] == 1)
		{
			ESKF(KF, H, y, &acc[i * 3 * n], &P[i * 3 * n], V, Q, ABias, GBias, applyZUPT[i * n], applyZARU[i * n], applyGPS[i * n], applyAltitude[i * n], Fx, Xx, handle);
			correct(P, V, Q, ABias, GBias, KF);
		}
		else
		{
			covariancePropagate(handle,  KF,  H, &acc[i*3*n], &Q[i*3*n],&ABias[i*3*n], Fx);
		}
		pos[i * 3] = P[0];
		pos[i * 3 + 1] = P[1];
		pos[i * 3 + 2] = P[2];
		var[i * 3] = KF->PKF[to_idx(1, 1, 15)];
		var[i * 3 + 1] = KF->PKF[to_idx(2, 2, 15)];
		var[i * 3 + 2] = KF->PKF[to_idx(3, 3, 15)];
	}
}

//smoother function
__global__ void pos_kernel(float* position1, float* position2, float* var1, float* var2,
	float* pos, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < 3 * n) {
		pos[3*i] = (var2[i] * position1[i] + var1[i] * position2[i]) / (var1[i] + var2[i]);
		pos[3 * i+1] = (var2[3 * i + 1] * position1[3 * i + 1] + var1[3 * i + 1] * position2[3 * i + 1]) / (var1[3 * i + 1] + var2[3 * i + 1]);
		pos[3 * i+2] = (var2[3 * i + 2] * position1[3 * i + 2] + var1[3 * i + 2] * position2[3 * i + 2]) / (var1[3 * i + 2] + var2[3 * i + 2]);
    }
}