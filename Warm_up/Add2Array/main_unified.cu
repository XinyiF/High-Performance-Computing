#include "vadd.cuh"
#include <iostream>
using namespace std;
int main(int argc, const char *argv[]) {
  string type;
  if (argc > 1) {
    type = string(argv[1]);
  }
  unsigned int n = atoi(type.c_str());
  float *dA, *dB;
  hipMallocManaged((float **)&dA, n * sizeof(float));
  hipMallocManaged((float **)&dB, n * sizeof(float));
  for (unsigned int i = 0; i < n; i++) {
    dA[i] = 0.5;
    dB[i] = 0.5;
  }

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  vadd<<<(n + 1023) / 1024, 1024>>>(dA, dB, n);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  std::printf("%f\n", ms / 1000);
  std::printf("%f\n", dB[0]);
  std::printf("%f\n", dB[n - 1]);
  hipFree(dA);
  hipFree(dB);

  return 0;
}
