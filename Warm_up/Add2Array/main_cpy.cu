#include "vadd.cuh"
#include <iostream>
using namespace std;

int main(int argc, const char *argv[]) {
  string type;
  if (argc > 1) {
    type = string(argv[1]);
  }
  unsigned int n = atoi(type.c_str());
  float *a = new float[n];
  float *b = new float[n];
  float *dA, *dB;
  hipMalloc((void **)&dA, n * sizeof(float));
  hipMalloc((void **)&dB, n * sizeof(float));
  for (unsigned int i = 0; i < n; i++) {
    a[i] = 0.5;
    b[i] = 0.5;
  }
  hipMemcpy(dA, a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dB, b, n * sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  vadd<<<(n + 511) / 512, 512>>>(dA, dB, n);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);

  hipMemcpy(a, dA, n * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(b, dB, n * sizeof(float), hipMemcpyDeviceToHost);
  printf("%f\n", ms / 1000);
  printf("%f\n", b[0]);
  printf("%f\n", b[n - 1]);

  delete[] a;
  delete[] b;
  hipFree(dA);
  hipFree(dB);

  return 0;
}
