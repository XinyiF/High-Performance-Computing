
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void vadd(const float *a, float *b, unsigned int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    b[i] = a[i] + b[i];
  }
}
