#include "reduce.cuh"
#include <iostream>
using namespace std;

int main(int argc, const char *argv[]) {
  string n, Threads;
  if (argc > 1) {
    n = string(argv[1]);
    Threads = string(argv[2]);
  }
  unsigned int N = atoi(n.c_str());
  unsigned int threads_per_block = atoi(Threads.c_str());

  int *arr = new int[N];
  for (unsigned int i = 0; i < N; i++) {
    arr[i] = 1;
  }
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  int sum = reduce(arr, N, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);

  cout << sum << endl;
  cout << ms << endl;
  delete[] arr;
  return 0;
}