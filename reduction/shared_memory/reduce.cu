#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include <iostream>
using namespace std;
__global__ void reduce_kernel(const int *g_idata, int *g_odata,
                              unsigned int n) {
  extern __shared__ int data[];
  unsigned int shared_tid = threadIdx.x;
  unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_tid < n) {
    data[shared_tid] = g_idata[global_tid];
  } else {
    data[shared_tid] = 0;
  }
  __syncthreads();
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (shared_tid < s) {
      data[shared_tid] += data[shared_tid + s];
    }
    __syncthreads();
  }
  g_odata[blockIdx.x] = data[0];
}

__host__ int reduce(const int *arr, unsigned int N,
                    unsigned int threads_per_block) {
  int *g_odata, *g_idata;
  int res = 0;
  int size = N;
  hipMalloc((void **)&g_idata, size * sizeof(int));
  hipMemcpy(g_idata, arr, size * sizeof(int), hipMemcpyHostToDevice);
  while (size > 1) {
    hipMalloc((void **)&g_odata, (size + threads_per_block - 1) /
                                      threads_per_block * sizeof(int));
    reduce_kernel<<<(size + threads_per_block - 1) / threads_per_block,
                    threads_per_block, threads_per_block * sizeof(int)>>>(
        g_idata, g_odata, size);

    size = (size + threads_per_block - 1) / threads_per_block;
    int *sum = new int[size];
    hipMemcpy(sum, g_odata, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(g_odata);
    hipFree(g_idata);
    hipMalloc((void **)&g_idata, size * sizeof(int));
    hipMemcpy(g_idata, sum, size * sizeof(int), hipMemcpyHostToDevice);
    res = sum[0];
    free(sum);
  }
  return res;
  hipDeviceSynchronize();
}