#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
using namespace std;

int main(int argc, const char *argv[]) {
  string N;
  if (argc > 1) {
    N = string(argv[1]);
  }
  unsigned int n = atoi(N.c_str());
  thrust::host_vector<int> H(n);
  for (unsigned int i = 0; i < n; i++) {
    H[i] = 1;
  }
  thrust::device_vector<int> D = H;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  int sum = thrust::reduce(D.begin(), D.end());
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  cout << sum << endl;
  cout << ms << endl;
  return 0;
}