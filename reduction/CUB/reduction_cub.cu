#include <hipcub/hipcub.hpp>
#include <cub/device/device_reduce.cuh>
#include <cub/util_allocator.cuh>
#include <iostream>
using namespace hipcub;
using namespace std;
CachingDeviceAllocator g_allocator(true);

int main(int argc, const char *argv[]) {
  string N, Threads;
  if (argc > 1) {
    N = string(argv[1]);
  }
  unsigned int n = atoi(N.c_str());
  int *h_in = new int[n];
  for (unsigned int i = 0; i < n; i++) {
    h_in[i] = 1;
  }
  // Set up device arrays
  int *d_in = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_in, sizeof(int) * n));
  // Initialize device input
  CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * n, hipMemcpyHostToDevice));
  // Setup device output array
  int *d_sum = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_sum, sizeof(int) * 1));
  // Request and allocate temporary storage
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CubDebugExit(
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  // Do the actual reduce operation
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  CubDebugExit(
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  int gpu_sum;
  CubDebugExit(
      hipMemcpy(&gpu_sum, d_sum, sizeof(int) * 1, hipMemcpyDeviceToHost));
  // Check for correctness
  cout << gpu_sum << endl;
  cout << ms << endl;

  // Cleanup
  if (d_in)
    CubDebugExit(g_allocator.DeviceFree(d_in));
  if (d_sum)
    CubDebugExit(g_allocator.DeviceFree(d_sum));
  if (d_temp_storage)
    CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
  delete[] h_in;

  return 0;
}
