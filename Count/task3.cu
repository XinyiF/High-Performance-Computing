#include "count.cuh"
#include <cstdlib>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <time.h>
using namespace std;

int main(int argc, const char *argv[]) {
  string N;
  if (argc > 1) {
    N = string(argv[1]);
  }
  int n = atoi(N.c_str());
  thrust::host_vector<int> H(n);
  srand((unsigned)time(NULL));
  for (int i = 0; i < n; i++) {
    H[i] = (rand() % 101);
  }
  thrust::device_vector<int> d_in = H;
  thrust::device_vector<int> values(n);
  thrust::device_vector<int> counts(n);
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  count(d_in, values, counts);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  cout << values[values.size() - 1] << endl;
  cout << counts[counts.size() - 1] << endl;
  cout << ms << endl;
  return 0;
}
