#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
using namespace std;

int main(int argc, const char *argv[]) {
  string N;
  if (argc > 1) {
    N = string(argv[1]);
  }
  unsigned int n = atoi(N.c_str());
  thrust::host_vector<float> H(n);
  for (unsigned int i = 0; i < n; i++) {
    H[i] = 1;
  }
  thrust::device_vector<float> D = H;
  thrust::device_vector<float> res(n);
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::exclusive_scan(D.begin(), D.end(), res.begin());
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  cout << res[n - 1] << endl;
  cout << ms << endl;
  return 0;
}