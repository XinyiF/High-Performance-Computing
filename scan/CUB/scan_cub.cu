#include <cub/device/device_scan.cuh>
#include <cub/util_allocator.cuh>
#include <iostream>
using namespace hipcub;
using namespace std;
CachingDeviceAllocator g_allocator(true);

int main(int argc, const char *argv[]) {
  string N;
  if (argc > 1) {
    N = string(argv[1]);
  }
  int n = atoi(N.c_str());
  float *h_in = new float[n];
  float *cpu_out = new float[n];
  for (int i = 0; i < n; i++) {
    h_in[i] = 1.0;
    cpu_out[i] = 0;
  }
  // Set up device arrays
  float *d_in = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_in, sizeof(float) * n));
  // Initialize device input
  CubDebugExit(
      hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));
  // Setup device output array
  float *d_out = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_out, sizeof(float) * n));
  // Request and allocate temporary storage
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                        d_in, d_out, n));

  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  // Do the actual reduce operation
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                             d_in, d_out, n));
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  CubDebugExit(
      hipMemcpy(cpu_out, d_out, sizeof(float) * n, hipMemcpyDeviceToHost));
  // Check for correctness
  cout << cpu_out[n - 1] << endl;
  cout << ms << endl;
  // Cleanup
  if (d_in)
    CubDebugExit(g_allocator.DeviceFree(d_in));
  if (d_out)
    CubDebugExit(g_allocator.DeviceFree(d_out));
  if (d_temp_storage)
    CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
  delete[] h_in;
  delete[] cpu_out;
  return 0;
}
