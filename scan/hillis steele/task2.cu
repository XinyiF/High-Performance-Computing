#include "scan.cuh"
#include <iostream>
using namespace std;

int main(int argc, const char *argv[]) {
  string N, Threads;
  if (argc > 1) {
    N = string(argv[1]);
  }
  unsigned int n = atoi(N.c_str());
  float *in = new float[n];
  float *out = new float[n];
  for (unsigned int i = 0; i < n; i++) {
    in[i] = 1;
    out[i] = 0;
  }
  unsigned int threads_per_block = 1024;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  scan(in, out, n, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  cout << out[n - 1] << endl;
  cout << ms << endl;
  delete[] out;
  delete[] in;

  return 0;
}
