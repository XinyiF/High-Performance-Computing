#include "hip/hip_runtime.h"
#include "scan.cuh"
#include <iostream>
#include <math.h>
using namespace std;
__global__ void scan_kernel(float *g_odata, float *g_idata, unsigned int n,
                            float *last_ele) {
  extern volatile __shared__ float data[];
  int thid = threadIdx.x;
  int g_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int pout = 0, pin = 1;
  data[thid] = (thid == 0) ? 0 : g_idata[g_tid - 1];
  __syncthreads();
  for (int offset = 1; offset < blockDim.x; offset *= 2) {
    pout = 1 - pout;
    pin = 1 - pout;
    if (thid >= offset)
      data[pout * blockDim.x + thid] = data[pin * blockDim.x + thid] +
                                       data[pin * blockDim.x + thid - offset];
    else
      data[pout * blockDim.x + thid] = data[pin * blockDim.x + thid];
    __syncthreads();
  }
  if (g_tid < n) {
    g_odata[g_tid] = data[pout * blockDim.x + thid];
    if (thid == blockDim.x - 1) {
      last_ele[blockIdx.x] = data[thid] + g_idata[g_tid];
    }
  }
}
__global__ void helper_kernel(float *g_odata, unsigned int n,
                              unsigned int threads_per_block, float *last_ele) {
  extern __shared__ float temp[];
  int s_tid = threadIdx.x;
  int g_tid = blockIdx.x * blockDim.x + s_tid;
  if (g_tid < n) {
    temp[s_tid] = g_odata[g_tid];
  } else {
    temp[s_tid] = 0;
  }
  __syncthreads();
  if (blockIdx.x > 0) {
    for (int i = 0; i < blockIdx.x; i++) {
      temp[s_tid] += last_ele[i];
    }
    __syncthreads();
  }
  if (g_tid < n) {
    g_odata[g_tid] = temp[s_tid];
  }
}

__host__ void scan(const float *in, float *out, unsigned int n,
                   unsigned int threads_per_block) {
  float *g_odata, *g_idata, *last_ele;
  hipMalloc((void **)&g_idata, n * sizeof(float));
  hipMemcpy(g_idata, in, n * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)&g_odata, n * sizeof(float));
  hipMalloc((void **)&last_ele,
             (n + threads_per_block - 1) / threads_per_block * sizeof(float));
  scan_kernel<<<(n + threads_per_block - 1) / threads_per_block,
                threads_per_block, 2 * threads_per_block * sizeof(float)>>>(
      g_odata, g_idata, n, last_ele);
  if (n > threads_per_block) {
    helper_kernel<<<(n + threads_per_block - 1) / threads_per_block,
                    threads_per_block, threads_per_block * sizeof(float)>>>(
        g_odata, n, threads_per_block, last_ele);
  }
  hipMemcpy(out, g_odata, n * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(g_idata);
  hipFree(g_odata);
  hipFree(last_ele);
  hipDeviceSynchronize();
}
