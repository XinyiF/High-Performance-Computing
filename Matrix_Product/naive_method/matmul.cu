#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <iostream>
using namespace std;
__global__ void matmul_kernel(const float *A, const float *B, float *C,
                              size_t n) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n * n) {
    size_t row = i / n;
    size_t col = i % n;
    C[i]=0;
    for (size_t r = 0; r < n; r++) {
      C[i] += A[row * n + r] * B[r * n + col];
    }
  }
}
void matmul(const float *A, const float *B, float *C, size_t n,
            unsigned int threads_per_block) {
  matmul_kernel<<<((n * n) + threads_per_block - 1) / threads_per_block,
                  threads_per_block>>>(A, B, C, n);
  hipDeviceSynchronize();
}

