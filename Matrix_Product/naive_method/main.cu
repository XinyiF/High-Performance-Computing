#include "matmul.cuh"
#include <iostream>
using namespace std;
int main(int argc, const char *argv[]) {
  string type, type1;
  if (argc > 1) {
    type = string(argv[1]);
    type1 = string(argv[2]);
  }
  size_t n = atoi(type.c_str());
  unsigned int threads_per_block = atoi(type1.c_str());
  float *A, *B, *C;
  hipMallocManaged((float **)&A, n * n * sizeof(float));
  hipMallocManaged((float **)&B, n * n * sizeof(float));
  hipMallocManaged((float **)&C, n * n * sizeof(float));
  for (unsigned int i = 0; i < n * n; i++) {
    A[i] = 0.5;
    B[i] = 0.5;
  }
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  matmul(A, B, C, n, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Get the elapsed time in milliseconds
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  cout << C[n * n - 1] << endl;
  cout << ms << endl;
  hipFree(A);
  hipFree(B);
  hipFree(C);

  return 0;
}

