#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <iostream>
using namespace std;
__global__ void matmul_kernel(const float *A, const float *B, float *C,
                              unsigned int n) {
  // tile size is blockDim.x*blockDim.y
  unsigned int global_tid = (blockIdx.y * blockDim.y + threadIdx.y) * n +
                            blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int block_tid = threadIdx.y * blockDim.x + threadIdx.x;
  unsigned int a_begin =
      blockIdx.y * blockDim.y * n; // the begin element of sub A
  unsigned int a_end = a_begin + n - 1;
  unsigned int b_begin = blockIdx.x * blockDim.x;
  float csub = 0;
  extern __shared__ float data[];
  for (unsigned int a = a_begin, b = b_begin; a < a_end;
       a += blockDim.x, b += blockDim.y * n) {
    // load subA and subB, each thread load an element of two sub matrix
    // if the index out of boundary, fill with 0
    if (a + threadIdx.y * n + threadIdx.x < n * n) {
      data[threadIdx.y * blockDim.y + threadIdx.x] =
          A[a + threadIdx.y * n + threadIdx.x];
    } else {
      data[threadIdx.y * blockDim.y + threadIdx.x] = 0;
    }
    if (b + threadIdx.y * n + threadIdx.x < n * n) {
      data[blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x] =
          B[b + threadIdx.y * n + threadIdx.x];
    } else {
      data[blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x] =
          0;
    }
    __syncthreads();
    // compute element of C of current thread
    for (unsigned int j = 0; j < blockDim.x; j++) {
      csub += data[threadIdx.y * blockDim.y + j] *
              data[blockDim.x * blockDim.y + j * blockDim.y + threadIdx.x];
    }
    __syncthreads();
  }
  if (global_tid < n * n) {
    C[global_tid] = csub;
  }
}

__host__ void matmul(const float *A, const float *B, float *C, unsigned int n,
                     unsigned int block_dim) {

  dim3 dimBlock(block_dim, block_dim);
  dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x,
               (n + dimBlock.y - 1) / dimBlock.y);
  matmul_kernel<<<dimGrid, dimBlock,
                  (2 * block_dim * block_dim) * sizeof(float)>>>(A, B, C, n);
  hipDeviceSynchronize();
}
