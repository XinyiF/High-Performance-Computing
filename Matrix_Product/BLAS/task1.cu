#include "mmul.h"
#include <hipblas.h>
#include <iostream>
using namespace std;
int main(int argc, const char *argv[]) {
  string N, r;
  if (argc > 1) {
    N = string(argv[1]);
    r = string(argv[2]);
  }
  int n = atoi(N.c_str());
  int n_test = atoi(r.c_str());
  float *A, *B, *C;
  hipMallocManaged((float **)&A, n * n * sizeof(float));
  hipMallocManaged((float **)&B, n * n * sizeof(float));
  hipMallocManaged((float **)&C, n * n * sizeof(float));
  for (unsigned int i = 0; i < n * n; i++) {
    A[i] = i;
    B[i] = i;
    C[i] = 0;
  }

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float avg = 0;
  for (int i = 0; i < n_test; i++) {
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    mmul(handle, A, B, C, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    avg += ms;
  }
  cout << avg / n_test << endl;

  // Destroy the handle
  hipblasDestroy(handle);

  hipFree(A);
  hipFree(B);
  hipFree(C);

  return 0;
}