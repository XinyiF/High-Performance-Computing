#include "mmul.h"
#include <hipblas.h>
#include <iostream>

void mmul(hipblasHandle_t handle, const float *A, const float *B, float *C,
          int n) {
  int lda = n, ldb = n, ldc = n;
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;

  // Do the actual multiplication
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A, lda, B, ldb,
              beta, C, ldc);
  hipDeviceSynchronize();
}